#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <math.h>
#include <string.h>
#include <time.h>

#define REAL double

// CUDA error check
#define cudaCheckError() { \
    hipError_t e=hipGetLastError(); \
    if(e!=hipSuccess) { \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
}

REAL find_max_error(REAL* x, int n) {
    REAL max_error = 0.0;
    for (int i = 0; i < n; i++) {
        REAL error = fabs(x[i] - 1.0);  // Assumes the expected solution is all ones
        if (error > max_error) {
            max_error = error;
        }
    }
    return max_error;
}

// Function to allocate memory and initialize the matrix and vectors
void generate_random_system(REAL** A, REAL** b, REAL** x, int n) {
    // Seed the random number generator to get different results each time
    srand(time(NULL));

    // Allocate memory for A, b, x
    *A = new REAL[n * n];
    *b = new REAL[n];
    *x = new REAL[n]; // This will be the solution vector; initialized later

    // Fill the matrix A and vector b
    for (int i = 0; i < n; i++) {
        (*b)[i] = 0.0; // Initialize b[i] to zero for accumulation
        for (int j = 0; j < n; j++) {
            if (i == j) {
                (*A)[i * n + j] = n / 10.0;
            } else {
                (*A)[i * n + j] = (REAL)rand() / RAND_MAX; // Random double between 0.0 and 1.0
            }
            (*b)[i] += (*A)[i * n + j];
        }
    }

    // Optionally initialize x to some default values
    for (int i = 0; i < n; i++) {
        (*x)[i] = 0.0; // Not necessary for solving, but good for initialization
    }
}

// Function to read matrix A and vector b from a file
void read_system(const char* filename, REAL** A, REAL** b, REAL** x, int* n) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    // Read the dimension of the matrix
    if (fscanf(file, "%d", n) != 1) {
        fprintf(stderr, "Invalid matrix file format\n");
        fclose(file);
        exit(EXIT_FAILURE);
    }

    // Allocate memory for A, b, x
    *A = new REAL[*n * *n];
    *b = new REAL[*n];
    *x = new REAL[*n];

    // Read the matrix A and vector b
    for (int i = 0; i < *n; i++) {
        for (int j = 0; j < *n; j++) {
            if (fscanf(file, "%lf", &(*A)[i * *n + j]) != 1) {
                fprintf(stderr, "Invalid matrix file format while reading A[%d][%d]\n", i, j);
                fclose(file);
                exit(EXIT_FAILURE);
            }
        }
        if (fscanf(file, "%lf", &(*b)[i]) != 1) {
            fprintf(stderr, "Invalid matrix file format while reading b[%d]\n", i);
            fclose(file);
            exit(EXIT_FAILURE);
        }
    }

    // Optionally initialize x to some default values (e.g., zeros)
    for (int i = 0; i < *n; i++) {
        (*x)[i] = 0.0; // Initialize solution vector to zero
    }

    fclose(file);
}

__device__ double atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void gaussian_elimination_kernel(REAL *A, REAL *b, int n, int pivot) {
    int row = threadIdx.x + pivot + 1;

    if (row < n) {
        REAL coeff = A[row * n + pivot] / A[pivot * n + pivot];
        for (int col = pivot; col < n; col++) {
            A[row * n + col] -= coeff * A[pivot * n + col];
        }
        b[row] -= coeff * b[pivot];
    }
}


// Kernel for backward substitution
__global__ void back_substitution_kernel(REAL *A, REAL *b, REAL *x, int n) {
    int row = n - blockIdx.x - 1;

    REAL sum = 0.0;
    for (int col = row + 1; col < n; col++) {
        sum += A[row * n + col] * x[col];
    }
    x[row] = (b[row] - sum) / A[row * n + row];
}

// Timer macro definitions using CUDA events
float cuda_timer_start, cuda_timer_stop;
hipEvent_t start, stop;

#define START_TIMER() { \
    hipEventCreate(&start); \
    hipEventCreate(&stop); \
    hipEventRecord(start); \
}

#define STOP_TIMER() ({ \
    hipEventRecord(stop); \
    hipEventSynchronize(stop); \
    float milliseconds = 0; \
    hipEventElapsedTime(&milliseconds, start, stop); \
    hipEventDestroy(start); \
    hipEventDestroy(stop); \
    milliseconds / 1000.0; \
})

#define GET_TIMER() (cuda_timer_start / 1000.0) // Return in seconds

// Function to print matrices (host side)
void print_matrix(REAL *mat, int rows, int cols) {
    for (int row = 0; row < rows; row++) {
        for (int col = 0; col < cols; col++) {
            printf("%8.1e ", mat[row * cols + col]);
        }
        printf("\n");
    }
}

int main(int argc, char *argv[]) {
    int n;          // Matrix size
    bool debug_mode = false;
    bool triangular_mode = false;
    
    // Parse command line arguments
    int c;
    while ((c = getopt(argc, argv, "dt")) != -1) {
        switch (c) {
        case 'd':
            debug_mode = true;
            break;
        case 't':
            triangular_mode = true;
            break;
        default:
            printf("Usage: %s [-dt] <file|size>\n", argv[0]);
            exit(EXIT_FAILURE);
        }
    }
    if (optind != argc - 1) {
        printf("Usage: %s [-dt] <file|size>\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    REAL *A, *b, *x; // Host pointers
    REAL *d_A, *d_b, *d_x; // Device pointers

    // Determine if input is a file or a matrix size
    char* input = argv[optind];
    long int size = strtol(input, NULL, 10);
    START_TIMER();
    if (size == 0) {
        read_system(input, &A, &b, &x, &n);
    } else {
        n = (int)size;
        generate_random_system(&A, &b, &x, n);
    }
    float init_time = STOP_TIMER();

    // Allocate device memory
    hipMalloc(&d_A, n * n * sizeof(REAL));
    hipMalloc(&d_b, n * sizeof(REAL));
    hipMalloc(&d_x, n * sizeof(REAL));
    cudaCheckError();

    // Copy data from host to device
    hipMemcpy(d_A, A, n * n * sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(REAL), hipMemcpyHostToDevice);
    cudaCheckError();

    if (debug_mode) {
        printf("Original A = \n");
        print_matrix(A, n, n);
        printf("Original b = \n");
        print_matrix(b, n, 1);
    }

    // Perform Gaussian elimination
    START_TIMER();
    if (!triangular_mode) {
        for (int pivot = 0; pivot < n - 1; pivot++) {
            int threadsPerBlock = 256;
            int numBlocks = (n - pivot - 1 + threadsPerBlock - 1) / threadsPerBlock;
            gaussian_elimination_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_b, n, pivot);

            hipDeviceSynchronize(); // Ensure completion before moving to the next pivot
            cudaCheckError();

            // Optionally: Copy back A and b to host to check the intermediate state
            hipMemcpy(A, d_A, n * n * sizeof(REAL), hipMemcpyDeviceToHost);
            hipMemcpy(b, d_b, n * sizeof(REAL), hipMemcpyDeviceToHost);
        }
    }
    float gaus_time = STOP_TIMER();


    // Synchronize
    hipDeviceSynchronize();

    // Perform back substitution
    START_TIMER();
    for (int i = n - 1; i >= 0; i--) {
        REAL sum = 0;
        for (int j = i + 1; j < n; j++) {
            sum += A[i * n + j] * x[j];
        }
        x[i] = (b[i] - sum) / A[i * n + i];
    }
    float bsub_time = STOP_TIMER();

    if (debug_mode) {
        printf("Triangular A = \n");
        print_matrix(A, n, n);
        printf("Updated b = \n");
        print_matrix(b, n, 1);
        printf("Solution x = \n");
        print_matrix(x, n, 1);
    }

    // Compute the maximum error
    REAL max_error = find_max_error(x, n);

    // Print results
    printf("Nthreads=%2d  ERR=%8.1e  INIT: %8.4fs  GAUS: %8.4fs  BSUB: %8.4fs\n",
       1, max_error, init_time, gaus_time, bsub_time);

    // Clean up and exit
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);
    delete[] A;
    delete[] b;
    delete[] x;

    return EXIT_SUCCESS;
}
